#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <time.h>
#include <math.h>
using namespace std;

#define blocks_num 384
#define thread_per_block 1024

// function to check the num if prime can call from host and device
bool Is_Prime(unsigned long long int num)
{
	for (unsigned long long int i = 2; i <= sqrtf(num); ++i)
	{
		if (num % i == 0)
			return false;
	}
	return true;
}

// Create a kernel to check number if prime
__global__ void Check_Prime(unsigned long long int *d_number,bool *d_out,int *d_iteration)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int loop_begin = index * (*d_iteration);
	int loop_end = (index + 1)* (*d_iteration);
	if(index == 0)
		loop_begin +=2;
	for(long long i = loop_begin; i<= loop_end ; i++)
	{
		if (*d_number % i == 0)
		{
			*d_out = false;
			return;
		}
		else
		{
			*d_out = true;
		}
			
	}		
}

void main()
{
	float CPU_TIME;
	float GPU_TIME;

	cout << "\t\t\t*** CUDA TASK ***\n\t\t\t==================\n\n";
	cout << "Checking Numbers ... :\n----------------------\n";
	
	// host variables
	unsigned long long int number = 100000000000000003;
    bool *out;
	int iteration_per_thread = int(sqrtf(number)/(blocks_num * thread_per_block));

	// device var
	unsigned long long int *d_number;
	bool *d_out;
	int *d_iteration;

	// allocate device data
	hipMalloc((void **)&d_number, sizeof(unsigned long long int));
	hipMalloc((void **)&d_out, sizeof(bool));
	hipMalloc((void **)&d_iteration, sizeof(int));

	// copy data from host to device 
	hipMemcpy(d_number, &number, sizeof(unsigned long long int), hipMemcpyHostToDevice);
	hipMemcpy(d_iteration, &iteration_per_thread, sizeof(int), hipMemcpyHostToDevice);

	hipEvent_t start, stop;       // define  2 events
	hipEventCreate(&start);       // create start event
	hipEventCreate(&stop);		   // create stop event	
	hipEventRecord(start, 0);     // begin start event

	// call check_prime kernal
	Check_Prime <<< blocks_num, thread_per_block >>> (d_number, d_out,d_iteration);

	hipEventRecord(stop, 0);      // begin stop event   
	hipEventSynchronize(stop);
	hipEventElapsedTime(&GPU_TIME, start, stop);   // calculate execution time
	
	// destroy 2 events
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	// copy data back from device to host
	hipMemcpy(out, d_out, sizeof(bool), hipMemcpyDeviceToHost);
	
	// print GPU data
	cout << "the number = "<<number<<" is ";
	if(*out == true)
		cout << "prime\n";
	else 
		cout << "not prime\n";  
	cout << "GPU Time = " << GPU_TIME << endl << "--------------------\n\n";

	// sequential code  
	unsigned long long int cpu_start = clock();      // cpu start time 
	
	cout << "the number = "<<number<<" is ";
	if (Is_Prime(number))
		cout << "prime\n";
	else
		cout << "not prime\n";  

	unsigned long long int cpu_stop = clock();       // cpu stop time
	CPU_TIME = float(cpu_stop - cpu_start);          // cpu execution time
	cout << "CPU Time = " << CPU_TIME << endl << "--------------------\n";  // print CPU data

	// free allocated data on device
	hipFree(d_number);
	hipFree(d_out);	
}